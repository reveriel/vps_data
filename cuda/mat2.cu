#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

// thread block size
#define BLOCK_SIZE 16



// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}


// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);






// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
/*
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);


    hipError_t err = hipDeviceSynchronize();
    printf("Run kernel: %s\n", hipGetErrorString(err));

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

*/
void MatMul(const Matrix A, const Matrix B, Matrix C) { 
  // Load A and B to device memory 
  Matrix d_A; 
  d_A.width = d_A.stride = A.width; 
  d_A.height = A.height; 
  size_t size = A.width * A.height * sizeof(float); 
  hipError_t err = hipMalloc(&d_A.elements, size); 
  printf("CUDA malloc A: %s\n",hipGetErrorString(err)); 
  err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice); 
  printf("Copy A to device: %s\n",hipGetErrorString(err)); 

  Matrix d_B; 
  d_B.width = d_B.stride = B.width; 
  d_B.height = B.height; 
  size = B.width * B.height * sizeof(float); 
  err = hipMalloc(&d_B.elements, size); 
  printf("CUDA malloc B: %s\n",hipGetErrorString(err));
  err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
  printf("Copy B to device: %s\n",hipGetErrorString(err)); 

  // Allocate C in device memory 
  Matrix d_C; 
  d_C.width = d_C.stride = C.width; 
  d_C.height = C.height; 
  size = C.width * C.height * sizeof(float); 
  err = hipMalloc(&d_C.elements, size); 
  printf("CUDA malloc C: %s\n",hipGetErrorString(err));

  // Invoke kernel 
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y); 
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C); 
    err = hipDeviceSynchronize();
    printf("Run kernel: %s\n", hipGetErrorString(err));

  // Read C from device memory 
  err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost); 
  printf("Copy C off of device: %s\n",hipGetErrorString(err));

  // Free device memory
  hipFree(d_A.elements); 
  hipFree(d_B.elements); 
  hipFree(d_C.elements); 
}

// Matrix multiplication kernel called by MatMul()
/*
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}
*/

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) { 
  // Block row and column 
  int blockRow = blockIdx.y; 
  int blockCol = blockIdx.x; 

  // Each thread block computes one sub-matrix Csub of C
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol); 

  // Each thread computes one element of Csub 
  // by accumulating results into Cvalue 
  float Cvalue = 0.0; 

  // Thread row and column within Csub 
  int row = threadIdx.y; 
  int col = threadIdx.x; 

  // Loop over all the sub-matrices of A and B that are 
  // required to compute Csub 
  // Multiply each pair of sub-matrices together 
  // and accumulate the results 
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
    // Get sub-matrix Asub of A 
    Matrix Asub = GetSubMatrix(A, blockRow, m); 

    // Get sub-matrix Bsub of B 
    Matrix Bsub = GetSubMatrix(B, m, blockCol); 

    // Shared memory used to store Asub and Bsub respectively 
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE]; 
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE]; 

    // Load Asub and Bsub from device memory to shared memory 
    // Each thread loads one element of each sub-matrix 
    As[row][col] = GetElement(Asub, row, col); 
    Bs[row][col] = GetElement(Bsub, row, col); 

    // Synchronize to make sure the sub-matrices are loaded 
    // before starting the computation 
    __syncthreads(); 

    // Multiply Asub and Bsub together 
    for (int e = 0; e < BLOCK_SIZE; ++e) 
      Cvalue += As[row][e] * Bs[e][col];
 
    // Synchronize to make sure that the preceding 
    // computation is done before loading two new 
    // sub-matrices of A and B in the next iteration 
    __syncthreads();  
  }

  // Write Csub to device memory 
  // Each thread writes one element 
  SetElement(Csub, row, col, Cvalue); 
}
void fill_Matrix(Matrix A)
{
    for (int i = 0; i < A.height ; i++) {
        for (int j = 0; j < A.width; j++) {
            A.elements[i * A.width + j] = rand() / (float)RAND_MAX * 10;
        }
    }
}

void print_Matrix(Matrix A)
{
    for (int i = 0; i < A.height; i++) {
        for (int j = 0; j < A.width; j++) {
            printf("%4.1f ", A.elements[i * A.width + j]);
        }
        printf("\n");
    }
}



int main(void)
{

    srand(time(0));
    int n = 1 << 3;
    Matrix A, B, C;
    A.width = A.height = n;
    A.elements = (float *)malloc(sizeof(float) * n * n);
    B.width = B.height = n;
    B.elements = (float *)malloc(sizeof(float) * n * n);
    C.width = C.height = n;
    C.elements = (float *)malloc(sizeof(float) * n * n);

    fill_Matrix(A);
    print_Matrix(A);
    printf("\n");

    fill_Matrix(B);
    print_Matrix(B);
    printf("\n");

    MatMul(A, B, C);

    print_Matrix(C);

}
